#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include <math.h>
#include "utils.h"

#define BLOCK_SIZE 32

/* https://stackoverflow.com/a/14038590/5832371
 * CUDA GPU error checking.
*/
inline void gpuAssert(hipError_t code, const char *file, int line, 
                      bool abort=true)
{
  if (code != hipSuccess) 
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", 
            hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
//#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
extern "C" void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }


double presidual(const long N, const double *u, const double *f)
{ //Compute ||Au - f|| under the frobenius norm
    double ihsq = N*N;
    double resid = 0.0;
    #pragma omp parallel for collapse(2) reduction(+:resid) 
    for (int j = 1; j < N-1; j++)
    {
        for (int i = 1; i < N-1; i++)
        {
            double v = ihsq * (- u[(i-1)+j*N] - u[i+(j-1)*N]
                               + 4*u[i+j*N]
                               - u[(i+1)+j*N] - u[i+(j+1)*N] )
                               - f[i+j*N];
            resid += v*v;
        }
    }
    return sqrt(resid);
}

/*
 * CPU Jacobi Poisson Step:
 * Computes one step of u[i,j] = (1/4)*(h^2 f[i,j] + u0[i-1,j]  + u0[i,j-1]  +
 * u0[i+1,j]  + u0[i,j+1]) for all 1 <= i, j < N-1.
 */
void jacobi_step_cpu(double *u, const double *u0, const double *f, const long N)
{
  double h = 1.0 / (double)N;
  #pragma omp parallel for collapse(2)
  for (long i = 1; i < N-1; i++)
  {
    for (long j = 1; j < N-1; j++)
    {
      u[i*N + j] = 0.25 * ( h*h*f[i*N + j] + u0[(i-1)*N + j] + 
                                             u0[i*N + (j-1)] + 
                                             u0[(i+1)*N + j] + 
                                             u0[i*N + (j+1)] ); 
    }
  }
}

/*
 * GPU Jacobi Poisson Step:
 * Computes one step of u[i,j] = (1/4)*(h^2 f[i,j] + u0[i-1,j]  + u0[i,j-1]  +
 * u0[i+1,j]  + u0[i,j+1]) for idx*BLOCK_SIZE <= i < (idx+1)*BLOCK_SIZE and
 * jdx*BLOCK_SIZE <= j < (jdx+1)*BLOCK_SIZE.
 */
__global__ void jacobi_step_gpu(
                double *u, const double *u0, const double *f, 
                const long N)
{
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x + 1;
  int jdx = (blockIdx.y) * blockDim.y + threadIdx.y + 1;
  double h = 1.0 / (double)N;
  u[idx*N + jdx] = 0.25 * ( h*h*f[idx*N + jdx] + u0[(idx-1)*N + jdx] + 
                                                 u0[idx*N + (jdx-1)] + 
                                                 u0[(idx+1)*N + jdx] + 
                                                 u0[idx*N + (jdx+1)] );
}
/*
 * Having learned safer GPU error handling practices, I try to be better here
 * code quality wise than my other two cuda functions. Also helps that I
 * understand this code MUCH more than whatever black magic that reduce method
 * is.
*/
int main(int argc, char** argv) 
{
  printf("Jacobi iteration with Cuda vs. CPU\n");

  const long N = 1<<10;
  const long N_grid = N+2; // Including ghost points
  const long MAX_ITERATESM1 = 1000;
  Timer t;

  // Malloc structures. Note we leave room for ghost points.
  double *f  = (double*) malloc(N_grid*N_grid*sizeof(double));
  double *u0 = (double*) malloc(N_grid*N_grid*sizeof(double));
  double *u  = (double*) malloc(N_grid*N_grid*sizeof(double));

  /* BEGIN CPU JACOBI POISSON */

  //Initialize vectors.  
  for (long i = 0; i < N_grid*N_grid; i++) { f[i] = 1; u0[i] = u[i] = 0; }

  printf("Initial residue: %.4e\n", presidual(N_grid, u, f));
  t.tic();

  for (int k = 0; k < MAX_ITERATESM1; k += 2)
  {
    jacobi_step_cpu(u, u0, f, N_grid);
    jacobi_step_cpu(u0, u, f, N_grid);
  }
  jacobi_step_cpu(u, u0, f, N_grid);

  double time = t.toc();
  printf("CPU computation: %.4f seconds: Final residue: %.4e\n", 
         time, presidual(N_grid, u, f) );

  // Reinitialize arrays
  for (long i = 0; i < N_grid*N_grid; i++) { f[i] = 1; u0[i] = u[i] = 0; }

  /* END CPU JACOBI POISSON */ /* BEGIN GPU JACOBI POISSON */

  // Allocate vectors onto GPU and transfer host data to device.
  double *f_d, *u_d, *u0_d;
  gpuErrchk( 
    hipMalloc(&f_d, N_grid*N_grid*sizeof(double)) 
  );
  gpuErrchk( 
    hipMemcpy(f_d, f, N_grid*N_grid*sizeof(double), hipMemcpyHostToDevice) 
  );
  gpuErrchk( 
    hipMalloc(&u_d, N_grid*N_grid*sizeof(double)) 
  );
  gpuErrchk( 
    hipMemcpy(u_d, u, N_grid*N_grid*sizeof(double), hipMemcpyHostToDevice) 
  );
  gpuErrchk( 
    hipMalloc(&u0_d, N_grid*N_grid*sizeof(double)) 
  );
  gpuErrchk( 
    hipMemcpy(u0_d, u0, N_grid*N_grid*sizeof(double), hipMemcpyHostToDevice) 
  );
  hipDeviceSynchronize();

  // Warp dimension calculation: code directly from cuda C programmers guide
  // Divide N into BLOCK_SIZE pieces, overshoot if not divisible.
  int GRID_SIZE = 0; // For some reason this breaks when BLOCK_SIZE defined.
  if ( N % BLOCK_SIZE == 0) { GRID_SIZE = N / BLOCK_SIZE; }
  else { GRID_SIZE = (N / BLOCK_SIZE) + 1; }

  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim(GRID_SIZE, GRID_SIZE); // From lecture9/filter.cu

  t.tic();

  for (long k = 0; k < MAX_ITERATESM1; k += 2)
  {
    jacobi_step_gpu<<<gridDim, blockDim>>>(u_d, u0_d, f_d, N_grid);
    jacobi_step_gpu<<<gridDim, blockDim>>>(u0_d, u_d, f_d, N_grid);
  }
  jacobi_step_gpu<<<gridDim, blockDim>>>(u_d, u0_d, f_d, N_grid);

  gpuErrchk( 
    hipMemcpy(u, u_d, N_grid*N_grid*sizeof(double), hipMemcpyDeviceToHost) 
  );
  hipDeviceSynchronize();

  time = t.toc();
  printf("GPU computation: %.4f seconds: Final residue: %.4e\n", 
         time, presidual(N_grid, u, f) );

  /* END GPU JACOBI POISSON */

  free(f); free(u0); free(u);
  gpuErrchk( hipFree(f_d) );
  gpuErrchk( hipFree(u_d) );
  gpuErrchk( hipFree(u0_d) );
}
